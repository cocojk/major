#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using CUDA's memory copy API to transfer data to and from the
 * device. In this case, hipMalloc is used to allocate memory on the GPU and
 * hipMemcpy is used to transfer the contents of host memory to an array
 * allocated using hipMalloc.
 */

int main(int argc, char **argv)
{

    // memory size
    unsigned int isize;
    unsigned int nbytes;
    
    // get the memory size as a command line argument
    if (argc == 2)
    {
        // its default size is MB
        nbytes = atoi(argv[1]) * (1<<20);
        isize = nbytes>>2;        
    }
    else 
    {
        printf("specify the memory size \n");
        exit(0);
    }

    
    // set up device
    int dev = 0;
    
    // return : hipError_t 
    // parameter : int device (device on which the active host thread should execute the device code)
    // records device as the device on which the active host thread executes the device code 
    CHECK(hipSetDevice(dev));


    // get device information
    hipDeviceProp_t deviceProp;
    // return : hipError_t
    // parameter : hipDeviceProp_t* prop , int device 
    // returns hipGetDeviceProperties of device 
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    // print the current information 
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size nbyte %5.2fMB\n", dev,
           deviceProp.name, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    float *h_a = (float *)malloc(nbytes);

    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize the host memory
    for(unsigned int i = 0; i < isize; i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_a));
    free(h_a);

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destroys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
