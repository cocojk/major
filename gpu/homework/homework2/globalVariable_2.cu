#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;


// print the current value of the global variable and increase by 2.0 
__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    // alter the value
    devData += 2.0f;
}

int main(void)
{
    // initialize the global variable
    float value = 3.14f;
    float* devPtr;

    // get the device global variable address 
    // return : hipError_t 
    // parameter : void** devPtr, const char* symbol 
    // return in (*devPtr) the address of symbol (symbol) on the devce 
    CHECK(hipGetSymbolAddress((void**)&devPtr,devData));    

    
    // transfer data from the host to the device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(devPtr, &value, sizeof(float),hipMemcpyHostToDevice));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // transfer data from the device to the host
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(&value, devPtr, sizeof(float),hipMemcpyDeviceToHost));
    printf("Host:   the value changed by the kernel to %f\n", value);

    

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destroys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
