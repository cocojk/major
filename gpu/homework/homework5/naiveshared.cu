#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

#define BLOCK_SIZE 32

// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : double* A (input matrix)
//             double* B (input matrix)
//             double* C (output matrix, C= AB)
//             int  nx (size of the matrix row & column)
__global__ void matrixMul (double* A, double* B, double* C, int size)
{

    // shared memory area , we assign the same size of current block size 
    __shared__ double subA [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double subB [BLOCK_SIZE][BLOCK_SIZE];


    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get the current matrix index to be calculated
    int idx = blockIdx.x*BLOCK_SIZE + tx;
    int idy = blockIdx.y*BLOCK_SIZE + ty;
    int currentVal = 0;

    // we calculate the matrix by partitioning the problem into smaller problem. each time we perform the subMatrix multiplication one by one. 
    // the size of smaller problem is BLOCK_SIZE 
    for (int i = 0; i < (size + BLOCK_SIZE -1) /BLOCK_SIZE ; i++)
    {
        // each thread in a block load the data from global memory to the shared memory corresponding their matrix index 
        if ( idx < size && idy < size)
        {
            subA[ty][tx] = A[idy*size + i*BLOCK_SIZE + tx];
            subB[ty][tx] = B[(i*BLOCK_SIZE+ty)*size + idx];
        }
        else
        {
            subA[ty][tx] = 0;
            subB[ty][tx] = 0;
        }

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();

        // calculate subMatrix multiplication 
        for (int j = 0; j < BLOCK_SIZE; j++)
            currentVal += subA[ty][j] * subB[j][tx];

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();
    }

    // finally update the corresponding value to the proper location
    if (idx < size && idx < size)
        C[idy*size + idx] = currentVal;
}

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, double **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    double *A = (double *)malloc(sizeof(double) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

// printf the partial matrix 
// return : void
// parameter : double * M - input matrix , int nrows - number of row in matrix, int ncols - number of column in matrix, int max_row - the max row to be printed, int max_col - the max col to be printed 
void print_partial_matrix(double *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}




int main (int argc, char ** argv) {

    double *A;
    double *B;
    double *C;


    int nx = 1024;
    int ny = 1024;
    int size = nx * ny;
    const int BufferSize = size * sizeof(double);

    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);

    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(nx, ny, &A);
    int trueBNnz = generate_random_dense_matrix(nx, ny, &B);


    C = (double*)malloc(BufferSize);



    double* d_A;
    double* d_B;
    double* d_C;

    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void**)&d_A,BufferSize));
    CHECK(hipMalloc((void**)&d_B,BufferSize));
    CHECK(hipMalloc((void**)&d_C,BufferSize));

    // fills the first (count) bytes of the memory area pointed to by (devPtr) with the constant byte value (value) 
    // return : hipError_t 
    // parameter : void* devPtr, int value, size_t count 
    CHECK(hipMemset(d_C,0,BufferSize));

    // copy data from host to device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_A,A,BufferSize,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,BufferSize,hipMemcpyHostToDevice));


    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (d_A,d_B,d_C,nx);

    CHECK(hipMemcpy (C,d_C,BufferSize,hipMemcpyDeviceToHost));



    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(A);
    free(B);
    free(C);

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}








