#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

#define BLOCK_SIZE 32

// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : double* A (input matrix)
//             double* B (input matrix)
//             double* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
__global__ void matrixMul (double* A, double* B, double* C, int size)
{

    // shared memory area , we assign the same size of current block size 
    __shared__ double subA [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double subB [BLOCK_SIZE][BLOCK_SIZE];

    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    
    // get the current matrix index to be calculated
    int idx = blockIdx.x*BLOCK_SIZE + tx;
    int idy = blockIdx.y*BLOCK_SIZE + ty;
    int currentVal = 0;

    // we calculate the matrix by partitioning the problem into smaller problem. each time we perform the subMatrix multiplication one by one. 
    // the size of smaller problem is BLOCK_SIZE 
    for (int i = 0; i < (size + BLOCK_SIZE -1) /BLOCK_SIZE ; i++)
    {
        
        // each thread in a block load the data from global memory to the shared memory corresponding their matrix index 
        if ( idx < size && idy < size)
        {
            subA[ty][tx] = A[idy*size + i*BLOCK_SIZE + tx];
            subB[ty][tx] = B[(i*BLOCK_SIZE+ty)*size + idx];
        }
        else
        {
            subA[ty][tx] = 0;
            subB[ty][tx] = 0;
        }

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();

        // calculate subMatrix multiplication 
        for (int j = 0; j < BLOCK_SIZE; j++)
            currentVal += subA[ty][j] * subB[j][tx];

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();
    }

    // finally update the corresponding value to the proper location
    if (idx < size && idx < size)
        C[idy*size + idx] = currentVal;
}

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, double *outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double *curr = outA + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0)
            {
                totalNnz++;
            }
        }
    }

    return totalNnz;
}

// printf the partial matrix 
// return : void
// parameter : double * M - input matrix , int nrows - number of row in matrix, int ncols - number of column in matrix, int max_row - the max row to be printed, int max_col - the max col to be printed 
void print_partial_matrix(double *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}




int main (int argc, char ** argv) {

    double *A;
    double *B;
    double *C;


    int nx = 1024;
    int ny = 1024;
    int size = nx * ny;
    const int BufferSize = size * sizeof(double);

    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);

    // allocate unified memory 
    // return hipError_t 
    // parameter : T** devPtr, size_t size, unsigned int flags 
    // allocate memory that will be automatically managed by the unified memory system
    CHECK(hipMallocManaged((void**)&A,BufferSize));
    CHECK(hipMallocManaged((void**)&B,BufferSize));
    CHECK(hipMallocManaged((void**)&C,BufferSize));


    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(nx, ny, A);
    int trueBNnz = generate_random_dense_matrix(nx, ny, B);
    
    


    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (A,B,C,nx);


    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());



    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(A));
    CHECK(hipFree(B));
    CHECK(hipFree(C));
    
    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}








