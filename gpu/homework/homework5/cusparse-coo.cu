#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, double **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    double *A = (double *)malloc(sizeof(double) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}



// printf the partial matrix 
// return : void
// parameter : double * M - input matrix , int nrows - number of row in matrix, int ncols - number of column in matrix, int max_row - the max row to be printed, int max_col - the max col to be printed 
void print_partial_matrix(double *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}

int main(int argc, char **argv)
{
    double *A, *dA;
    double *B, *dB;
    double *C, *dC;
    int *dANnzPerRow;
    double *dCooValA;
    
    int *dCsrRowPtrA;
    int *dCooRowIndA;
    int *dCooColIndA;

    int totalANnz;
    double alpha = 1.0;
    double beta = 4.0;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t Adescr = 0;

    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(M, N, &A);
    int trueBNnz = generate_random_dense_matrix(N, M, &B);
    C = (double *)malloc(sizeof(double) * M * M);

    // print partial matrix
    printf("A:\n");
    print_partial_matrix(A, M, N, 10, 10);
    printf("B:\n");
    print_partial_matrix(B, N, M, 10, 10);

    // Create the cuSPARSE handle
    // return : hipsparseStatus_t 
    // parameter : hipsparseHandle_t * handle 
    // initialize the cuSPARSE library and creates a handle on the cuSPARSE context. it allocates hardware resources necessary for accessing the GPU
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Allocate device memory for vectors and the dense form of the matrix A
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void **)&dA, sizeof(double) * M * N));
    CHECK(hipMalloc((void **)&dB, sizeof(double) * N * M));
    CHECK(hipMalloc((void **)&dC, sizeof(double) * M * M));
    CHECK(hipMalloc((void **)&dANnzPerRow, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    // return : hipsparseStatus_t
    // parameter : cudsparseMatDescr_t * descrA
    // the function initialize the matrix descriptor. 
    // It sets the fields MatrixType and IndexBase to the default values HIPSPARSE_MATRIX_TYPE_GENERAL and HIPSPARSE_INDEX_BASE_ZERO 
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&Adescr));
    
    // set the MatrixType field of the matrix descriptor descrA 
    // return : hipsparseStatus_t 
    // parameter : hipsparseMatDescr_t descrA, hipsparseMatrixType_t type 
    CHECK_CUSPARSE(hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    
    // set the IndexBase field of the matrix descriptor descrA 
    // return : hipsparseStatus_t 
    // parameter : hipsparseMatDescr_t descrA , hipsparseIndexBase_t base 
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO));

    // Transfer the input vectors and dense matrix A to the device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(dA, A, sizeof(double) * M * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(double) * N * M, hipMemcpyHostToDevice));
    CHECK(hipMemset((void*)dC, 0, sizeof(double) * M * M));

    // Compute the number of non-zero elements in A and compute the number of non-zero elements in A per row 
    // return : hipsparseStatus_t 
    // parameter : hipsparseHandle_t handle , hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const double *A, int lda, int* nnzPerRowColumn, int* nnzTotalDevHostPtr 
    // input : handle - handle to the cuSPARSE library context 
    //         dirA   - direction that specifies whether to count nonzero elements by HIPSPARSE_DIRECTION_ROW or by HIPSPARSE_DIRECTION_COLUMN
    //         m      - number of rows of matrix A 
    //         n      - number of columns of matrix A
    //         descrA - the descriptor of matrix A 
    //         A      - array of dimensions (lda,n)
    //         lda    - leading dimension of dense array A 
    // output: nnzPerRowColumn - array of size m 
    //       : nnzTotalDevHostPtr - total number of nonzero elements in device or host memory 
    CHECK_CUSPARSE(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, Adescr,
                                dA, M, dANnzPerRow, &totalANnz));

    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }

    // Allocate device memory to store the sparse coo & csr representation of A
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void **)&dCooValA, sizeof(double) * totalANnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCooColIndA, sizeof(int) * totalANnz));
    CHECK(hipMalloc((void **)&dCooRowIndA, sizeof(int) * totalANnz));

    // there is no coo function that can convert A from a dense formattint to a coo formatiing. so first conver to csr format 
    // Convert A from a dense formatting to a CSR formatting, using the GPU
    // return : hipsparseStatus_t 
    // parameter : hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t  descA, const double *A, int lda, const int *nnzPerCol, double *cscValA, int* cscRowIndA,int* cscColPtrA 
    CHECK_CUSPARSE(hipsparseDdense2csr(handle, M, N, Adescr, dA, M, dANnzPerRow,
                                      dCooValA, dCsrRowPtrA, dCooColIndA));


    // conver csr format to coo format
    // return : hipsparseStatus_t 
    // parameter : hipsparseHandle_t handle, const int * csrRowPtr, int nnz, int m , int * cooRowInd, hipsparseIndexBase_t idxBase
    CHECK_CUSPARSE(hipsparseXcsr2coo(handle,dCsrRowPtrA,totalANnz,M,dCooRowIndA,HIPSPARSE_INDEX_BASE_ZERO));

    // Perform matrix-matrix multiplication with the CSR-formatted matrix A cause there is no coo matrix-matrix multiplication and matrix-vector multiplication  
    // parameter : hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int k, int nnz, const double *alpha, 
    //             const hipsparseMatDescr_t descrA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, 
    //             const double *B, int ldb, const double *beta, double *C, int ldc
    CHECK_CUSPARSE(hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M,
                                  M, N, totalANnz, &alpha, Adescr, dCooValA,
                                  dCsrRowPtrA, dCooColIndA, dB, N, &beta, dC,
                                  M));

    // Copy the result vector back to the host
    CHECK(hipMemcpy(C, dC, sizeof(double) * M * M, hipMemcpyDeviceToHost));

    // print the result partial matrix
    printf("C:\n");
    print_partial_matrix(C, M, M,10,10);

    // free the memory 
    free(A);
    free(B);
    free(C);

    CHECK(hipFree(dA));
    CHECK(hipFree(dB));
    CHECK(hipFree(dC));
    CHECK(hipFree(dANnzPerRow));
    CHECK(hipFree(dCooValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCooColIndA));
    CHECK(hipFree(dCooRowIndA));

    // releases the memory allocated for the matrix descriptor
    // return : hipsparseStatus_t 
    // parameter : cusparseDescr_t descrA 
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    
    // releases CPU-side resources used by the cuSPARSE library. 
    // return : hipsparseStatus_t 
    // parameter : hipsparseHandle_t handle
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
