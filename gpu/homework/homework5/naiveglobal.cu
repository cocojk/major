#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"


// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
__global__ void matrixMul (double *A, double *B, double *C, int nx)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // get the current index in matrix C
    unsigned int destIdx = iy * nx + ix;

    if (ix < nx && iy < nx)
    {
        for (int idx = 0; idx < nx; idx++)
        {
            C[destIdx] += A[iy*nx + idx]*B[idx*nx + ix];
            
        }
    }
    
}
   

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, double **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    double *A = (double *)malloc(sizeof(double) * M * N);
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
            int r = rand();
            double *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 100.0;
            }

            if (*curr != 0.0)
            {
                totalNnz++;
            }
        }
    }

    *outA = A;
    return totalNnz;
}

// printf the partial matrix 
// return : void
// parameter : double * M - input matrix , int nrows - number of row in matrix, int ncols - number of column in matrix, int max_row - the max row to be printed, int max_col - the max col to be printed 
void print_partial_matrix(double *M, int nrows, int ncols, int max_row,
        int max_col)
{
    int row, col;

    for (row = 0; row < max_row; row++)
    {
        for (col = 0; col < max_col; col++)
        {
            printf("%2.2f ", M[row * ncols + col]);
        }
        printf("...\n");
    }
    printf("...\n");
}



int main (int argc, char **argv)
{

    double* A;
    double* B;
    double* C;


    int nx = 1024;
    int ny = 1024;
    int dimx = 32;
    int dimy = 32;
    int size = nx * ny;
    const int BufferSize = size * sizeof(double);

    dim3 block(dimx,dimy);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);


    // Generate input
    srand(9384);
    int trueANnz = generate_random_dense_matrix(nx, ny, &A);
    int trueBNnz = generate_random_dense_matrix(nx, ny, &B);
    
    
    C = (double*)malloc(BufferSize);

    
    double* d_A;
    double* d_B;
    double* d_C;


    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void**)&d_A,BufferSize));
    CHECK(hipMalloc((void**)&d_B,BufferSize));
    CHECK(hipMalloc((void**)&d_C,BufferSize));
    
    // fills the first (count) bytes of the memory area pointed to by (devPtr) with the constant byte value (value) 
    // return : hipError_t 
    // parameter : void* devPtr, int value, size_t count 
    CHECK(hipMemset(d_C,0,BufferSize));

    // copy data from host to device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_A,A,BufferSize,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,BufferSize,hipMemcpyHostToDevice));
   
   
    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (d_A,d_B,d_C,nx);

    CHECK(hipMemcpy (C,d_C,BufferSize,hipMemcpyDeviceToHost));
    
       
   

    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    
    free(A);
    free(B);
    free(C);
    
    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}




