#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

#define BLOCK_SIZE 32

// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
__global__ void matrixMul (int* A, int* B, int* C, int size)
{

    // shared memory area , we assign the same size of current block size 
    __shared__ int subA [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int subB [BLOCK_SIZE][BLOCK_SIZE];

    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    
    // get the current matrix index to be calculated
    int idx = blockIdx.x*BLOCK_SIZE + tx;
    int idy = blockIdx.y*BLOCK_SIZE + ty;
    int currentVal = 0;

    // we calculate the matrix by partitioning the problem into smaller problem. each time we perform the subMatrix multiplication one by one. 
    // the size of smaller problem is BLOCK_SIZE 
    for (int i = 0; i < (size + BLOCK_SIZE -1) /BLOCK_SIZE ; i++)
    {
        
        // each thread in a block load the data from global memory to the shared memory corresponding their matrix index 
        if ( idx < size && idy < size)
        {
            subA[ty][tx] = A[idy*size + i*BLOCK_SIZE + tx];
            subB[ty][tx] = B[(i*BLOCK_SIZE+ty)*size + idx];
        }
        else
        {
            subA[ty][tx] = 0;
            subB[ty][tx] = 0;
        }

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();

        // calculate subMatrix multiplication 
        for (int j = 0; j < BLOCK_SIZE; j++)
            currentVal += subA[ty][j] * subB[j][tx];

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();
    }

    // finally update the corresponding value to the proper location
    if (idx < size && idx < size)
        C[idy*size + idx] = currentVal;
}

// host program to check the answer 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
// return : int (1 : correct 0: not correct)
int check (int* A, int* B, int* C, int * hostCheck,int size)
{
    // calculate the answer
    for (int col = 0; col < size; col++)
        for (int row = 0; row < size; row++)
        {
            int outidx = col*size  + row;
            for (int idx = 0; idx < size; idx++) 
            {
                hostCheck[outidx] += A[col*size + idx]*B[idx*size + row];
            }
        }


    // compare the answer
    for (int col = 0; col < size; col++)
        for (int row = 0; row < size; row++)
        {
            if (hostCheck[col*size + row] != C[col*size + row])
                return 0;
        }

    return 1;
}

int main (int argc, char ** argv) {

    int *A;
    int *B;
    int *C;
    int *hostCheck;


    int nx = 512;
    int ny = 512;
    int size = nx * ny;
    const int BufferSize = size * sizeof(int);

    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);

    // allocate unified memory 
    // return hipError_t 
    // parameter : T** devPtr, size_t size, unsigned int flags 
    // allocate memory that will be automatically managed by the unified memory system
    CHECK(hipMallocManaged((void**)&A,BufferSize));
    CHECK(hipMallocManaged((void**)&B,BufferSize));
    CHECK(hipMallocManaged((void**)&C,BufferSize));
    CHECK(hipMallocManaged((void**)&hostCheck,BufferSize));


    // init the value
    for (int i = 0; i < size; i++)
    {
        A[i] = i % 1000;
        B[i] = i % 1000;
        C[i] = 0;
        hostCheck[i] = 0;
    }


    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (A,B,C,nx);


    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());

    // check the answer
    if(!check(A,B,C,hostCheck,nx))
    {
        printf("not correct answer\n");
        exit(1);
    }
    else 
    {
        printf("correct answer\n");
    }


    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(A));
    CHECK(hipFree(B));
    CHECK(hipFree(C));
    CHECK(hipFree(hostCheck));
    
    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}








