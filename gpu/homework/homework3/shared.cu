#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

#define BLOCK_SIZE 1

// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
__global__ void matrixMul (int* A, int* B, int* C, int size)
{

    // shared memory area , we assign the same size of current block size 
    __shared__ int subA [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int subB [BLOCK_SIZE][BLOCK_SIZE];


    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get the current matrix index to be calculated
    int idx = blockIdx.x*BLOCK_SIZE + tx;
    int idy = blockIdx.y*BLOCK_SIZE + ty;
    int currentVal = 0;

    // we calculate the matrix by partitioning the problem into smaller problem. each time we perform the subMatrix multiplication one by one. 
    // the size of smaller problem is BLOCK_SIZE 
    for (int i = 0; i < (size + BLOCK_SIZE -1) /BLOCK_SIZE ; i++)
    {
        // each thread in a block load the data from global memory to the shared memory corresponding their matrix index 
        if ( idx < size && idy < size)
        {
            subA[ty][tx] = A[idy*size + i*BLOCK_SIZE + tx];
            subB[ty][tx] = B[(i*BLOCK_SIZE+ty)*size + idx];
        }
        else
        {
            subA[ty][tx] = 0;
            subB[ty][tx] = 0;
        }

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();

        // calculate subMatrix multiplication 
        for (int j = 0; j < BLOCK_SIZE; j++)
            currentVal += subA[ty][j] * subB[j][tx];

        // wait all of the thread in the block to ensure correct program answer.
        __syncthreads();
    }

    // finally update the corresponding value to the proper location
    if (idx < size && idx < size)
        C[idy*size + idx] = currentVal;
}

// host program to check the answer 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
// return : int (1 : correct 0: not correct)
int check (int* A, int* B, int* C, int * hostCheck,int size)
{

    // calculate the answer
    for (int col = 0; col < size; col++)
        for (int row = 0; row < size; row++)
        {
            int outidx = col*size  + row;
            for (int idx = 0; idx < size; idx++) 
            {
                hostCheck[outidx] += A[col*size + idx]*B[idx*size + row];
            }
        }


    // compare the answer
    for (int col = 0; col < size; col++)
        for (int row = 0; row < size; row++)
        {
            if (hostCheck[col*size + row] != C[col*size + row])
                return 0;
        }

    return 1;
}

int main (int argc, char ** argv) {

    int *A;
    int *B;
    int *C;
    int *hostCheck;


    int nx = 512;
    int ny = 512;
    int size = nx * ny;
    const int BufferSize = size * sizeof(int);

    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);

    A = (int*)malloc(BufferSize);
    B = (int*)malloc(BufferSize);
    C = (int*)malloc(BufferSize);
    hostCheck = (int*)malloc(BufferSize);

    // init the value
    for (int i = 0; i < size; i++)
    {
        A[i] = i % 1000;
        B[i] = i % 1000;
        C[i] = 0;
        hostCheck[i] = 0;
    }

    int* d_A;
    int* d_B;
    int* d_C;

    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void**)&d_A,BufferSize));
    CHECK(hipMalloc((void**)&d_B,BufferSize));
    CHECK(hipMalloc((void**)&d_C,BufferSize));

    // fills the first (count) bytes of the memory area pointed to by (devPtr) with the constant byte value (value) 
    // return : hipError_t 
    // parameter : void* devPtr, int value, size_t count 
    CHECK(hipMemset(d_C,0,sizeof(int)*size));

    // copy data from host to device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_A,A,BufferSize,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,BufferSize,hipMemcpyHostToDevice));


    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (d_A,d_B,d_C,nx);

    CHECK(hipMemcpy (C,d_C,BufferSize,hipMemcpyDeviceToHost));


    // check the answer
    if(!check(A,B,C,hostCheck,nx))
    {
        printf("not correct answer\n");
        exit(1);
    }
    else 
    {
        printf("correct answer\n");
    }

    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    free(A);
    free(B);
    free(C);
    free(hostCheck);

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}








