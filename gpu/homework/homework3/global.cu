#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"


// matrix multiplcation kernel function 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
__global__ void matrixMul (int *A, int *B, int *C, int nx)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // get the current index in matrix C
    unsigned int destIdx = iy * nx + ix;

    if (ix < nx && iy < nx)
    {
        for (int idx = 0; idx < nx; idx++)
        {
            C[destIdx] += A[iy*nx + idx]*B[idx*nx + ix];
            
        }
    }
    
}
   

// host program to check the answer 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix size)
// return : int (1 : correct 0: not correct)
int check (int* A, int* B, int* C, int * hostCheck,int size)
{
    
    // calculate the answer
    for (int col = 0; col < size; col++)
    {
        for (int row = 0; row < size; row++)
        {
            int outidx = col*size  + row;
            for (int idx = 0; idx < size; idx++) 
            {
                hostCheck[outidx] += A[col*size + idx]*B[idx*size + row];
            }
        }
    }


    // compare the answer
    for (int col = 0; col < size; col++)
        for (int row = 0; row < size; row++)
        {
            if (hostCheck[col*size + row] != C[col*size + row])
            {
                printf("[%d , %d] host : %d, device : %d\n",col,row,hostCheck[col*size + row],C[col*size + row]);
                return 0;
            }
        }

    return 1;
}


int main (int argc, char **argv)
{

    int* A;
    int* B;
    int* C;
    int* hostCheck;


    int nx = 512;
    int ny = 512;
    int dimx = 32;
    int dimy = 32;
    int size = nx * ny;
    const int BufferSize = size * sizeof(int);

    dim3 block(dimx,dimy);
    dim3 grid ( (nx + block.x -1)/block.x, (ny + block.y -1)/block.y);

    printf("grid [%d %d], block [%d %d] \n",grid.x,grid.y,block.x,block.y);
    A = (int*)malloc(BufferSize);
    B = (int*)malloc(BufferSize);
    C = (int*)malloc(BufferSize);
    hostCheck = (int*)malloc(BufferSize);

    // init the value 
    for (int i = 0; i < size; i++)
    {
        A[i] = i % 1000;
        B[i] = i % 1000;
        C[i] = 0;
        hostCheck[i] = 0;
    }

    int* d_A;
    int* d_B;
    int* d_C;


    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((void**)&d_A,BufferSize));
    CHECK(hipMalloc((void**)&d_B,BufferSize));
    CHECK(hipMalloc((void**)&d_C,BufferSize));
    
    // fills the first (count) bytes of the memory area pointed to by (devPtr) with the constant byte value (value) 
    // return : hipError_t 
    // parameter : void* devPtr, int value, size_t count 
    CHECK(hipMemset(d_C,0,BufferSize));

    // copy data from host to device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_A,A,BufferSize,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,BufferSize,hipMemcpyHostToDevice));
   
   
    // kernel function : run the matrixmultiplication
    matrixMul<<<grid,block>>> (d_A,d_B,d_C,nx);

    CHECK(hipMemcpy (C,d_C,BufferSize,hipMemcpyDeviceToHost));
    
       
    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());
   
    // check the answer 
    if(!check(A,B,C,hostCheck,nx))
    {
        printf("not correct answer\n");
        exit(1);
    }
    else 
    {
        printf("correct answer\n");
    }

    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    
    free(A);
    free(B);
    free(C);
    free(hostCheck);
    
    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());

    return 0;
}




