#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned reads on performance by
 * forcing misaligned reads to occur on a float*.
 */

// parameter : float * hostRef, float* gpuRef, const int N
// compare the data pointed by (hostRef) and the data pointed by (gpuRef). (N) is the number of compared data 
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

// parameter : float *ip, int size 
// init the data pointed by (ip). (size) is the number of inited data
void initialData(float *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 100.0f;
    }

    return;
}

// parameter : float *A, float *B ,float *C 
// skip the (offset), then add the data pointed by (A) to the data pointed by (B) and save that data to the (C) in the host  
void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[k] = A[idx] + B[idx];
    }
}

// parameter : float * A, float * B, float *C 
// skip the (offset), then add the data pointed by (A) to the data pointed by (B) and save that data to the (C) in the device 
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}


// parameter : float * A, float * B, float *C 
// skip the (offset), then add the data pointed by (A) to the data pointed by (B) and save that data to the (C) in the device 
__global__ void readOffset(float *A, float *B, float *C, const int n,
                           int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    
    // return : hipError_t 
    // parameter : int device (device on which the active host thread should execute the device code)
    // records device as the device on which the active host thread executes the device code 
    CHECK(hipSetDevice(dev));
    
    // get device information
    // return : hipError_t
    // parameter : hipDeviceProp_t* prop , int device 
    // returns hipGetDeviceProperties of device 
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    // get the offset from the argument 
    if (argc > 1) offset    = atoi(argv[1]);

    // get the blocksize from the argument
    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    //  summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    // allocate the device memory
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    //  kernel 1:
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup     <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    
    // return : hipError_t 
    // parameter : void 
    // return the last error 
    CHECK(hipGetLastError());

    iStart = seconds();
    readOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    
    // return : hipError_t 
    // parameter : void 
    // return the last error 
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    // return : hipError_t 
    // parameter : void* dst, const void* src, size_t count, enum hipMemcpyKind kind 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (dst), where (kind) is one of enum type
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);

    // free host and device memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
