#include "hip/hip_runtime.h"

#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>



// number of stream in the program 
#define NSTREAM 8


// host program to calculate the correct answer 
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix)
//             int  N (size of the matrix)
void sumArraysOnHost(int *A, int *B, int *C, const int N)
{
    // calculate the answer 
    for (int idx = 0; idx < N; idx++)
        C[idx] = A[idx] + B[idx];
}

// matrix addition kernel function 
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C)
//             int  N (size of the matrix )
__global__ void sumArrays(int *A, int *B, int *C, const int N)
{
    // calculate the unique threadId 
    // unique threadId can be used to determine the position of addition 
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int idx = iy * blockDim.x * gridDim.x + ix; 

    if (idx < N)
    {
      C[idx] = A[idx] + B[idx];
    }
}

// host and kernel result compare function
// parameter : int* hostRef (host result)
//             int* gpuRef (kernel result)
//             int  N (size of the matrix)
void checkResult(int *hostRef,int *gpuRef, const int N)
{
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        // if the result is not same 
        if (hostRef[i] != gpuRef[i]) 
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %d gpu %d at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    // if the result is same 
    if (match) printf("Arrays match.\n\n");
}

int main(int argc, char **argv)
{

    int dev = 0;
    
    hipDeviceProp_t deviceProp;
    
    // get device information 
    // return : hipError_t
    // parameter : hipDeviceProp_t* prop , int device 
    // returns hipGetDeviceProperties of device 
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    
    // return : hipError_t 
    // parameter : int device (device on which the active host thread should execute the device code)
    // records device as the device on which the active host thread executes the device code 
    CHECK(hipSetDevice(dev));


    // set up max connectioin
    setenv ("CUDA_DEVICE_MAX_CONNECTIONS", "4", 1);

    int nx = 8192;
    int ny = 8192;
    int dimx = 32;
    int dimy = 32;

    // each stream subset problem size 
    int subset = nx*ny/ NSTREAM;
    
    // kernel configuration
    // we just slice the x-axis to divide the problem
    dim3 block(dimx,dimy);
    dim3 grid((nx/NSTREAM+block.x-1)/block.x,(ny+block.y-1)/block.y);


    
    // malloc pinned host memory for async memcpy
    // return : hipError_t 
    // parameter : void** pHost, size_t size , unsigned int flags 
    // allocates (size) bytes of host memory that is page-locked and accessible to the device 
    int *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nx*ny*sizeof(int), hipHostMallocDefault));

    // init the data 
    for (int i = 0; i < (nx*ny); i++)
    {
        h_A[i] = i;
        h_B[i] = i;
        hostRef[i] = 0;
        gpuRef[i] = 0;
    }
    

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nx*ny);

    // malloc device global memory
    int *d_A, *d_B, *d_C;
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((int**)&d_A, nx*ny*sizeof(int)));
    CHECK(hipMalloc((int**)&d_B, nx*ny*sizeof(int)));
    CHECK(hipMalloc((int**)&d_C, nx*ny*sizeof(int)));




    // stream declare 
    hipStream_t stream[NSTREAM];

    // create a new asynchronous stream 
    // return : hipError_t 
    // parameter : hipStream_t* pStream 
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }


    // initiate all work on the device asynchronously in depth-first order
    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * subset;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], subset*sizeof(int),
                              hipMemcpyHostToDevice, stream[i]));
        CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], subset*sizeof(int),
                              hipMemcpyHostToDevice, stream[i]));
        sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset],
                &d_C[ioffset], subset);
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], subset*sizeof(int),
                              hipMemcpyDeviceToHost, stream[i]));
    }


    
    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());
    

    // check device results
    checkResult(hostRef, gpuRef, nx*ny);

    // free device global memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free page-locked memory 
    // return : hipError_t 
    // parameter : void* ptr 
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));


    // destroys and cleans up an asynchronous stream 
    // return : hipError_t 
    // parameter : hipStream_t stream  
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return(0);
}
