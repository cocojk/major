#include "hip/hip_runtime.h"

#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>



// number of stream in the program 
#define NSTREAM 8


// host program to calculate the correct answer 
// we assume that input matrix is always square
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix row&column)
void mulArraysOnHost(int *A, int *B, int *C, const int size)
{
    for (int col = 0; col < size; col++)
    {
        for (int row = 0; row < size; row++)
        {
            int outidx = col*size  + row;
            for (int idx = 0; idx < size; idx++) 
            {
                C[outidx] += A[col*size + idx]*B[idx*size + row];
            }
        }
    }
}

// matrix multiplcation kernel function 
// parameter : int* A (input matrix)
//             int* B (input matrix)
//             int* C (output matrix, C= AB)
//             int  nx (size of the matrix row)
//             int  ny (size of the matrix column)
__global__ void mulArrays(int *A, int *B, int *C, const int nx,const int ny)
{

    // calculate the unique threadId 
    // unique threadId can be used to determine the position of addition 
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;


    unsigned int idx = iy * blockDim.x * gridDim.x + ix; 
    
    // matrix multiplication 
    if (ix < nx && iy < ny)
    {
        for (int temp = 0 ; temp < nx; temp++)
        {
            C[idx] += A[iy*nx + temp]*B[temp*nx + ix];
        }
    }

}

// host and kernel result compare function
// parameter : int* hostRef (host result)
//             int* gpuRef (kernel result)
//             int  N (size of the matrix)
void checkResult(int *hostRef,int *gpuRef, const int N)
{
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        // if the result is not same 
        if (hostRef[i] != gpuRef[i]) 
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %d gpu %d at %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    // if the result is same 
    if (match) printf("Arrays match.\n\n");
}

int main(int argc, char **argv)
{

    int dev = 0;
    
    hipDeviceProp_t deviceProp;
    
    // get device information 
    // return : hipError_t
    // parameter : hipDeviceProp_t* prop , int device 
    // returns hipGetDeviceProperties of device 
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    
    // return : hipError_t 
    // parameter : int device (device on which the active host thread should execute the device code)
    // records device as the device on which the active host thread executes the device code 
    CHECK(hipSetDevice(dev));


    // set up max connectioin
    setenv ("CUDA_DEVICE_MAX_CONNECTIONS", "4", 1);

    int nx = 512;
    int ny = 512;
    int dimx = 32;
    int dimy = 32;

    // each stream subset problem size 
    int subset = nx*ny/ NSTREAM;
    
    
    // kernel configuration
    // we slice the y-axis to divide the problem 
    // to assign each stream 1/8 problem
    // we should divide the output matrix 1/8 (C=AB)
    // to get a 1/8 matrix, we need 1/8 A matrix and all B matrix 
    // ex) A (32,32) B (32,32)
    // to get a C (8,32) = A(8,32) * B(32,32) 
    dim3 block(dimx,dimy);
    dim3 grid((nx+block.x-1)/block.x,(ny/NSTREAM+block.y-1)/block.y);



    // malloc pinned host memory for async memcpy
    // return : hipError_t 
    // parameter : void** pHost, size_t size , unsigned int flags 
    // allocates (size) bytes of host memory that is page-locked and accessible to the device 
    int *h_A, *h_B, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&h_B, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nx*ny*sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nx*ny*sizeof(int), hipHostMallocDefault));

    // init the data 
    for (int i = 0; i < (nx*ny); i++)
    {
        h_A[i] = i;
        h_B[i] = i;
        hostRef[i] = 0;
        gpuRef[i] = 0;
    }


    // add vector at host side for result checks
    mulArraysOnHost(h_A, h_B, hostRef, nx);

    // malloc device global memory
    int *d_A, *d_B, *d_C;
    // return : hipError_t 
    // parameter : void** devPtr, size_t size 
    // allocates (size) bytes of lineare memory on the device and returns in (*devPtr) a pointer to the allocated memory 
    CHECK(hipMalloc((int**)&d_A, nx*ny*sizeof(int)));
    CHECK(hipMalloc((int**)&d_B, nx*ny*sizeof(int)));
    CHECK(hipMalloc((int**)&d_C, nx*ny*sizeof(int)));


    // first init the output matrix in device 
    CHECK(hipMemset(d_C,0,nx*ny*sizeof(int)));

    // we copy all data from host to device in matrix B
    CHECK(hipMemcpy(d_B,h_B,nx*nx*sizeof(int),hipMemcpyHostToDevice));


    // to correct answer, we need syn the previous hipMemcpy
    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());
    
    // stream declare 
    hipStream_t stream[NSTREAM];

    // create a new asynchronous stream 
    // return : hipError_t 
    // parameter : hipStream_t* pStream 
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }


    // breath first search step 1 
    // copy the A (1/8 size) matrix to the device 
    for (int i = 0; i < NSTREAM; ++i)
    {
        int ioffset = i * subset;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], subset*sizeof(int),
                    hipMemcpyHostToDevice, stream[i]));
    }

    
    // breath first search step 2
    // kernel launch 
    // each kernel calculate 1/8 subproblem 
    for (int i = 0; i < NSTREAM; i++)
    {
        int ioffset = i * subset;
        mulArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], d_B,
                &d_C[ioffset], nx,ny/NSTREAM);
    }

    // breath first search step 3 
    // get a result matrix C (1/8 size) from the device  
    for (int i = 0; i < NSTREAM; i++)
    {
        int ioffset = i * subset;
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], subset*sizeof(int),
                    hipMemcpyDeviceToHost, stream[i]));
    }



    // return : hipError_t 
    // parameter : void 
    // blocks until the device has completed all preceding requsted tasks.
    CHECK(hipDeviceSynchronize());


    // check device results
    checkResult(hostRef, gpuRef, nx*ny);

    // free device global memory
    // return : hipError_t 
    // parameter : void* devPtr 
    // free the memory space pointed to by devPtr 
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    // return : hipError_t 
    // parameter : void* ptr 
    CHECK(hipHostFree(h_A));
    CHECK(hipHostFree(h_B));
    CHECK(hipHostFree(hostRef));
    CHECK(hipHostFree(gpuRef));


    // destroys and cleans up an asynchronous stream 
    // return : hipError_t 
    // parameter : hipStream_t stream  
    for (int i = 0; i < NSTREAM; ++i)
    {
        CHECK(hipStreamDestroy(stream[i]));
    }

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destoys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return(0);
}
