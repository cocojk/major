#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>


void CHECK (hipError_t error)
{                     

    if (error != hipSuccess)
    {
        printf("Error : %s : %d, ",__FILE__,__LINE__);
        printf("code : %d, reason : %s\n",error,hipGetErrorString(error));
        exit(1);
    }
}
__global__ void VectorAdd (int* a, int* b, int* c,int nx)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy*nx + ix;
    c[idx] = a[idx] + b[idx];
}


double cpuSecond () {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main (int argc, char** argv) 
{
  //  double iStart = cpuSecond();

    const int nx = 8192;
    const int ny = 8192;
    int blockX =32;
    int blockY =32;
/*
    if (argc > 2 ) {
        blockX = atoi(argv[1]);
        blockY = atoi(argv[2]);
    }
    else 
    {
        exit(0);
    }
*/
    dim3 block (blockX,blockY);
    dim3 grid ( (nx + block.x -1)/block.x , (ny + block.y -1)/block.y);



    const int BufferSize = nx*ny*sizeof(int);

    int* A;
    int* B;
    int* Sum;

    A = (int*)malloc(BufferSize);
    B = (int*)malloc(BufferSize);
    Sum = (int*)malloc(BufferSize);

    int i = 0;

    for (int i = 0; i < (nx*ny); i++)
    {
        A[i] = i;
        B[i] = i;
        Sum[i] = 0;
    }

    int* d_A;
    int* d_B;
    int* d_Sum;

    CHECK(hipMalloc((void**)&d_A,BufferSize));
    CHECK(hipMalloc((void**)&d_B,BufferSize));
    CHECK(hipMalloc((void**)&d_Sum,BufferSize));

    CHECK(hipMemcpy(d_A,A,BufferSize,hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,B,BufferSize,hipMemcpyHostToDevice));

    VectorAdd<<<grid,block>>> (d_A,d_B,d_Sum,nx);


    CHECK(hipMemcpy (Sum,d_Sum,BufferSize,hipMemcpyDeviceToHost));

    for (i = 0; i < 5; i++) {
        printf("Result[%d] : %d\n",i,Sum[i]);}


    for (i = nx*ny-5; i < (nx*ny); i++) {
        printf("Result[%d] : %d\n",i,Sum[i]);}

    hipDeviceSynchronize();

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_Sum));

    free(A);
    free(B);
    free(Sum);

//    double iElaps = cpuSecond() - iStart;

//    printf("cpu time : %lf\n",iElaps);
    return 0;
}

