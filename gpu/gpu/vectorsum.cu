#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void VectorAdd (int* a, int* b, int* c)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
printf("blockIdx : %d, blockDim : %d\n",blockIdx.x,blockDim.x);
c[tid] = a[tid] + b[tid];
}

int main () 
{
const int size = 512*65535;
const int BufferSize = size*sizeof(int);

int* A;
int* B;
int* Sum;

A = (int*)malloc(BufferSize);
B = (int*)malloc(BufferSize);
Sum = (int*)malloc(BufferSize);

int i = 0;

for (int i = 0; i < size; i++)
{
A[i] = i;
B[i] = i;
Sum[i] = 0;
}

int* d_A;
int* d_B;
int* d_Sum;

hipMalloc((void**)&d_A,size*sizeof(int));
hipMalloc((void**)&d_B,size*sizeof(int));
hipMalloc((void**)&d_Sum,size*sizeof(int));

hipMemcpy(d_A,A,size*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_B,B,size*sizeof(int),hipMemcpyHostToDevice);

VectorAdd<<<65535,512>>> (d_A,d_B,d_Sum);

hipMemcpy (Sum,d_Sum,size*sizeof(int),hipMemcpyDeviceToHost);

for (i = 0; i < 5; i++) {
printf("Result[%d] : %d\n",i,Sum[i]);}


for (i = size-5; i < size; i++) {
printf("Result[%d] : %d\n",i,Sum[i]);}


hipFree(d_A);
hipFree(d_B);
hipFree(d_Sum);

free(A);
free(B);
free(Sum);

return 0;
}

