#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;


// print the current value of the global variable and increase by 2.0 
__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    // alter the value
    devData += 2.0f;
}

int main(void)
{
    // initialize the global variable
    float value = 3.14f;

    // return : hipError_t 
    // parameter : const char* symbol, const void* src, size_t count, size_t offset = 0, enum hipMemcpyKind kind = hipMemcpyHostToDevice 
    // copies (count) bytes from the memory area pointed by (src) to the memory area pointed to by (offset) bytes from the start of symbol (symbol) 
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // copy the global variable back to the host
    // return : hipError_t 
    // parameter : void* dst, const char* symbol, size_t count, size_t offset = 0, enum hipMemcpyKind kind = hipMemcpyDeviceToHost 
    // copies (count) bytes from the memory area pointed by (offset) bytes from the start of symbol (symbol) to the memory area pointed to by (dat)
    CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:   the value changed by the kernel to %f\n", value);

    

    // reset device
    // return : hipError_t 
    // parameter : void 
    // explicitly destroys and cleans up all resources associated with the current device in the current process 
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
